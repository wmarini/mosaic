#include "mosaic.h"
#include "mosaicexceptions.h"
#include "UtilNPP/Exceptions.h"
#include "UtilNPP/ImageIO.h"
#include "UtilNPP/ImagesCPU.h"
#include "UtilNPP/ImagesNPP.h"
#include <npp.h>
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include <hip/hip_runtime.h>
#include <filesystem>
#include <iostream>

namespace mosaic {

namespace {

template <typename T>
void PrintImageInfo(const T& image, const std::string& name = "")
{
    std::cout 
        << name << " image info:\n"
        << "  width: " << image.width() 
        << "\n  height: " << image.height()
        << "\n  pitch: " << image.pitch()
        << std::endl;
}

inline void HighPassBorderFilter(
        const Npp8u* pSrc, Npp32s nSrcStep, NppiSize oSrcSize, NppiPoint oSrcOffset, 
        Npp8u* pDst, Npp32s nDstStep, NppiSize oSizeROI)
{
    NPP_CHECK_NPP(nppiFilterHighPassBorder_8u_C1R(
        pSrc, nSrcStep, oSrcSize, oSrcOffset,
        pDst, nDstStep, oSizeROI, 
        NPP_MASK_SIZE_5_X_5, NPP_BORDER_REPLICATE));
}

} // namespace

void ProcessMosaic(
        const MosaicConfig& config,
        const std::string& in_image_fname,
        const std::string& out_image_fname)
{
    try {
        // declare a host image object for an 8-bit grayscale image
        npp::ImageCPU_8u_C1 oHostSrc;
        // load gray-scale image from disk
        npp::loadImage(in_image_fname, oHostSrc);
        std::cout << "Loaded image: " << in_image_fname << std::endl;
        PrintImageInfo(oHostSrc, in_image_fname);

        // declare a device image and copy construct from the host image,
        // i.e. upload host to device
        npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);
        PrintImageInfo(oDeviceSrc, "ImageNPP");

        // create struct with box-filter mask size
        NppiSize oMaskSize = {5, 5};

        NppiSize oSrcSize = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
        NppiPoint oSrcOffset = {0, 0};

        // create struct with ROI size
        NppiSize oSizeROI = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
        // allocate device image of appropriately reduced size
        npp::ImageNPP_8u_C1 oDeviceDst(oSizeROI.width, oSizeROI.height);

        // run box filter
        HighPassBorderFilter(
            oDeviceSrc.data(), oDeviceSrc.pitch(), oSrcSize, oSrcOffset,
            oDeviceDst.data(), oDeviceDst.pitch(), oSizeROI);

        // declare a host image for the result
        npp::ImageCPU_8u_C1 oHostDst(oDeviceDst.size());
        // and copy the device result data into it
        oDeviceDst.copyTo(oHostDst.data(), oHostDst.pitch());
        npp::saveImage(out_image_fname, oHostDst);
        PrintImageInfo(oHostDst, out_image_fname);
        std::cout << "Saved image: " << out_image_fname << std::endl;

        //-CHECK_CUDA(hipDeviceSynchronize());
        nppiFree(oDeviceSrc.data());
        nppiFree(oDeviceDst.data());
        hipDeviceReset();
    } catch (const npp::Exception& e) {
        throw mosaic::MosaicException(e.toString());
    }
}

}