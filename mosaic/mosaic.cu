#include "hip/hip_runtime.h"
#include "mosaic.h"
#include "UtilNPP/Exceptions.h"
#include "UtilNPP/ImageIO.h"
#include "UtilNPP/ImagesCPU.h"
#include "UtilNPP/ImagesNPP.h"
#include <npp.h>
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include <hip/hip_runtime.h>
#include <filesystem>
#include <iostream>

namespace mosaic {

namespace {

ImageList GetImageList(const std::string& images_dir)
{
    ImageList image_list;

    
    return image_list;
}

} // namespace

__host__
void ProcessMosaic(const MosaicConfig&& config)
{
    std::cout << "Images directory: " 
        << config.images_path << std::endl;
    auto image_list = GetImageList(config.images_path);
    std::cout << "# Images to be processed: " 
        << image_list.size() << std::endl;

    // declare a host image object for an 8-bit grayscale image
    npp::ImageCPU_8u_C1 oHostSrc;
    // load gray-scale image from disk
    npp::loadImage("/home/wmarini/projects/gpulab/game/mosaic/data/lenna-gray.png", oHostSrc);
    // declare a device image and copy construct from the host image,
    // i.e. upload host to device
    npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);

    // create struct with box-filter mask size
    NppiSize oMaskSize = {5, 5};

    NppiSize oSrcSize = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
    NppiPoint oSrcOffset = {0, 0};

    // create struct with ROI size
    NppiSize oSizeROI = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
    // allocate device image of appropriately reduced size
    npp::ImageNPP_8u_C1 oDeviceDst(oSizeROI.width, oSizeROI.height);
    // set anchor point inside the mask to (oMaskSize.width / 2,
    // oMaskSize.height / 2) It should round down when odd
    NppiPoint oAnchor = {oMaskSize.width / 2, oMaskSize.height / 2};

    // run box filter
    NPP_CHECK_NPP(nppiFilterBoxBorder_8u_C1R(
        oDeviceSrc.data(), oDeviceSrc.pitch(), oSrcSize, oSrcOffset,
        oDeviceDst.data(), oDeviceDst.pitch(), oSizeROI, oMaskSize, oAnchor,
        NPP_BORDER_REPLICATE));

    // declare a host image for the result
    npp::ImageCPU_8u_C1 oHostDst(oDeviceDst.size());
    // and copy the device result data into it
    oDeviceDst.copyTo(oHostDst.data(), oHostDst.pitch());
    std::string sResultFilename = "/home/wmarini/projects/gpulab/game/mosaic/data/lenna-gray-out.png";
    npp::saveImage(sResultFilename, oHostDst);
    std::cout << "Saved image: " << sResultFilename << std::endl;

    nppiFree(oDeviceSrc.data());
    nppiFree(oDeviceDst.data());
}

}