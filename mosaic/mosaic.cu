#include "hip/hip_runtime.h"
#include "mosaic.h"
#include "mosaicexceptions.h"
#include "UtilNPP/Exceptions.h"
#include "UtilNPP/ImageIO.h"
#include "UtilNPP/ImagesCPU.h"
#include "UtilNPP/ImagesNPP.h"
#include <npp.h>
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include <hip/hip_runtime.h>
#include <filesystem>
#include <iostream>

namespace mosaic {

namespace {

inline void HighPassBorderFilter(
        const Npp8u* pSrc, Npp32s nSrcStep, NppiSize oSrcSize, NppiPoint oSrcOffset, 
        Npp8u* pDst, Npp32s nDstStep, NppiSize oSizeROI)
{
    NPP_CHECK_NPP(nppiFilterHighPassBorder_8u_C1R(
        pSrc, nSrcStep, oSrcSize, oSrcOffset,
        pDst, nDstStep, oSizeROI, 
        NPP_MASK_SIZE_5_X_5, NPP_BORDER_REPLICATE));
}

} // namespace

__host__
void ProcessMosaic(
        const MosaicConfig& config,
        const std::string& in_image_fname,
        const std::string& out_image_fname)
{
    try {
        // declare a host image object for an 8-bit grayscale image
        npp::ImageCPU_8u_C1 oHostSrc;
        // load gray-scale image from disk
        npp::loadImage(in_image_fname, oHostSrc);
        // declare a device image and copy construct from the host image,
        // i.e. upload host to device
        npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);

        // create struct with box-filter mask size
        NppiSize oMaskSize = {5, 5};

        NppiSize oSrcSize = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
        NppiPoint oSrcOffset = {0, 0};

        // create struct with ROI size
        NppiSize oSizeROI = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
        // allocate device image of appropriately reduced size
        npp::ImageNPP_8u_C1 oDeviceDst(oSizeROI.width, oSizeROI.height);
        // set anchor point inside the mask to (oMaskSize.width / 2,
        // oMaskSize.height / 2) It should round down when odd
        NppiPoint oAnchor = {oMaskSize.width / 2, oMaskSize.height / 2};

        // run box filter
        HighPassBorderFilter(
            oDeviceSrc.data(), oDeviceSrc.pitch(), oSrcSize, oSrcOffset,
            oDeviceDst.data(), oDeviceDst.pitch(), oSizeROI);

        // declare a host image for the result
        npp::ImageCPU_8u_C1 oHostDst(oDeviceDst.size());
        // and copy the device result data into it
        oDeviceDst.copyTo(oHostDst.data(), oHostDst.pitch());
        npp::saveImage(out_image_fname, oHostDst);
        std::cout << "Saved image: " << out_image_fname << std::endl;

        nppiFree(oDeviceSrc.data());
        nppiFree(oDeviceDst.data());
        hipDeviceReset();
    } catch (const npp::Exception& e) {
        throw mosaic::MosaicException(e.toString());
    }
}

}