#include "mosaic.h"
#include <npp.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <filesystem>
#include <iostream>

namespace mosaic {

namespace {

ImageList GetImageList(const std::string& images_dir)
{
    ImageList image_list;

    
    return image_list;
}

} // namespace

__host__
void ProcessMosaic(const MosaicConfig&& config)
{
    std::cout << "Images directory: " 
        << config.images_path << std::endl;
    auto image_list = GetImageList(config.images_path);
    std::cout << "# Images to be processed: " 
        << image_list.size() << std::endl;

    using namespace cv;
    //Create an 8 bit single channel image
    Mat img = imread("data/lenna.png", IMREAD_GRAYSCALE );
    std::cout << img.size() << std::endl;
}

}