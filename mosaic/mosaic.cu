#include "mosaic.h"
#include <npp.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <filesystem>
#include <iostream>

namespace mosaic {

namespace {

ImageList GetImageList(const std::string& images_dir)
{
    ImageList image_list;

    
    return image_list;
}

} // namespace

__host__
void ProcessMosaic(const MosaicConfig&& config)
{
    std::cout << "Images directory: " 
        << config.images_path << std::endl;
    auto image_list = GetImageList(config.images_path);
    std::cout << "# Images to be processed: " 
        << image_list.size() << std::endl;

    using namespace cv;
    //Create an 8 bit single channel image
    Mat img = imread("data/lenna.png", IMREAD_GRAYSCALE );
    std::cout << img.size() << std::endl;
    const std::size_t img_size = img.total() * img.elemSize();

    Npp8u* dSrc;
    hipMalloc<Npp8u>(&dSrc, img_size);

    Npp8u* dDst;
    hipMalloc<Npp8u>(&dDst, img_size);

    hipMemcpy(dSrc, img.data, img_size, hipMemcpyHostToDevice);

    NppiSize size = {img.cols, img.rows};
    const Npp8u value = 150;

    nppiAddC_8u_C1IRSfs(value, dSrc, img.step, size, 1);

    hipMemcpy(img.data, dDst, img_size, hipMemcpyDeviceToHost);

    imwrite("data/lenna_out.png", img);

    hipFree(dSrc);
    hipFree(dDst);
}

}