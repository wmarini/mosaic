#include "mosaic.h"
#include <npp.h>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <iostream>

namespace mosaic {

namespace {

ImageList GetImageList(const std::string& images_dir)
{
    const std::filesystem::path images_path(images_dir);
    ImageList image_list;

    for (auto p : std::filesystem::directory_iterator(images_path)) {
        if (p.is_regular_file()) {
            std::cout << p.path().string() << std::endl;
            image_list.push_back(p.path().string());
        }
    }
    return image_list;
}

} // namespace

__host__
void ProcessMosaic(const MosaicConfig&& config)
{
    std::cout << "Images directory: " 
        << config.images_path << std::endl;
    auto image_list = GetImageList(config.images_path);
    std::cout << "# Images to be processed: " 
        << std::size(image_list) << std::endl;
}

}