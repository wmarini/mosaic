#include "common.h"
#include <hip/hip_runtime.h>
#include <npp.h>
#include <iostream>

namespace common {
    
__host__ void check(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " 
            << file << ':' << line 
            << " code=" << static_cast<unsigned int>(result) << ' '
            << hipGetErrorString(result) << " in function " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__host__ CudaVersion CUDAVersion()
{
    int n_devices;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&n_devices));
    if (n_devices > 0) {
        hipDeviceProp_t prop;
        CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0));
        return CudaVersion{prop.major,prop.minor};
    }
    return CudaVersion{0,0};
}

__host__ CudaVersion NPPVersion()
{
    const NppLibraryVersion* lib_ver = nppGetLibVersion();
    return (lib_ver != nullptr) ? 
                CudaVersion{lib_ver->major,lib_ver->minor} : 
                CudaVersion{0,0};
}

} // namespace common
